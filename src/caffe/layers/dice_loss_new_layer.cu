#include "hip/hip_runtime.h"
#include <algorithm>
#include <cfloat>
#include <vector>

#include "caffe/layers/dice_loss_new_layer.hpp"
#include "caffe/util/math_functions.hpp"


namespace caffe {

template <typename Dtype>
  void DiceLossNewLayer<Dtype>::Forward_gpu(
    const vector<Blob<Dtype>*>& bottom, const vector<Blob<Dtype>*>& top) {
  // Compute the loss (negative log likelihood)
  //const int count = bottom[0]->count();
    const int num = bottom[0]->shape(0);
    const int channel = bottom[0]->shape(1);
    const int ndata = bottom[0]->count(2);
  // Stable version of loss computation from input data
    const Dtype* input_data = bottom[0]->gpu_data();
    const Dtype* target = bottom[1]->gpu_data();
    Dtype temp_loss = (Dtype)0;
    int denominator;
    if (behavior&2) {
      for (int n=0; n < num; ++n) {
        for (int c=0; c< channel;++c) {
          Dtype PP=0, GG=0, PG=0;
          int start_index=n*ndata*channel+c*ndata;
	  caffe_gpu_dot(ndata, input_data + start_index, input_data + start_index, &PP);
	  caffe_gpu_dot(ndata, target + start_index, target + start_index, &GG);
	  caffe_gpu_dot(ndata, input_data + start_index, target + start_index, &PG);
          temp_loss+=(Dtype)(2.0)*PG/(PP + GG);
        }
      }
      denominator=num*channel;
    }
    else {
      for (int c=0; c < channel; ++c) {
        Dtype PP=0, GG=0, PG=0;
        for (int n=0; n < num; ++n) {
          Dtype PP_tmp=0, GG_tmp=0, PG_tmp=0;
          int start_index=n*ndata*channel+c*ndata;
	  caffe_gpu_dot(ndata, input_data + start_index, input_data + start_index, &PP_tmp);
	  PP += PP_tmp;
	  caffe_gpu_dot(ndata, target + start_index, target + start_index, &GG_tmp);
	  GG += GG_tmp;
	  caffe_gpu_dot(ndata, input_data + start_index, target + start_index, &PG_tmp);
	  PG += PG_tmp;
        }
        temp_loss+=(Dtype)(2.0)*PG/(PP + GG);
      }
      denominator=channel;

    }
    if (behavior&1) {
      top[0]->mutable_gpu_data()[0]=temp_loss/((Dtype)denominator);
    }
    else{
      top[0]->mutable_gpu_data()[0]=temp_loss;
    }
  }

template <typename Dtype>
__global__ void DiceLossNewBackward(const int n, const int start_index, const Dtype* input, const Dtype* target, Dtype PP, Dtype GG, Dtype PG, const int size, Dtype* bottom_diff) {
  CUDA_KERNEL_LOOP(index, n) {
    bottom_diff[start_index + index] =-(Dtype)(2.0) * (target[start_index + index] * (PP + GG) - ((Dtype)(2.0) * input[start_index + index] * PG))/((Dtype)(size) * (PP + GG) * (PP + GG));
  }
}

template <typename Dtype>
  void DiceLossNewLayer<Dtype>::Backward_gpu(
    const vector<Blob<Dtype>*>& top, const vector<bool>& propagate_down,
    const vector<Blob<Dtype>*>& bottom) {
    if (propagate_down[1]) {
      LOG(FATAL) << this->type()
      << " Layer cannot backpropagate to label inputs.";
    }
    if (propagate_down[0]) {
    // First, compute the diff
      const int num = bottom[0]->shape(0);
      const int channel = bottom[0]->shape(1);
      const int ndata = bottom[0]->count(2);
      Dtype* bottom_diff = bottom[0]->mutable_gpu_diff();
      const Dtype* input_data = bottom[0]->gpu_data();
      const Dtype* target = bottom[1]->gpu_data();
      if (behavior&2) {
        for (int n=0; n<num; ++n) {
          for (int c=0; c<channel; ++c) {
            Dtype PP=0, GG=0, PG=0;
            int start_index=n*ndata*channel+c*ndata;
	    caffe_gpu_dot(ndata, input_data + start_index, input_data + start_index, &PP);
	    caffe_gpu_dot(ndata, target + start_index, target + start_index, &GG);
	    caffe_gpu_dot(ndata, input_data + start_index, target + start_index, &PG);
	    if (behavior&1) {
	      DiceLossNewBackward<Dtype><<<CAFFE_GET_BLOCKS(ndata), CAFFE_CUDA_NUM_THREADS>>>(ndata, start_index, input_data, target, PP, GG, PG, num*channel, bottom_diff);
	    } else {
	      DiceLossNewBackward<Dtype><<<CAFFE_GET_BLOCKS(ndata), CAFFE_CUDA_NUM_THREADS>>>(ndata, start_index, input_data, target, PP, GG, PG, 1, bottom_diff);
	    }
          }
        }
      }
      else {
        for (int c=0; c<channel; ++c) {
          Dtype PP=0, GG=0, PG=0;
          for (int n=0; n<num; ++n) {
            Dtype PP_tmp=0, GG_tmp=0, PG_tmp=0;
            int start_index=n*ndata*channel+c*ndata;
	    caffe_gpu_dot(ndata, input_data + start_index, input_data + start_index, &PP_tmp);
	    PP += PP_tmp;
	    caffe_gpu_dot(ndata, target + start_index, target + start_index, &GG_tmp);
	    GG += GG_tmp;
	    caffe_gpu_dot(ndata, input_data + start_index, target + start_index, &PG_tmp);
	    PG += PG_tmp;
          }
          for (int n=0; n<num; ++n) {
            int start_index=n*ndata*channel+c*ndata;
	    if (behavior&1) {
	      DiceLossNewBackward<Dtype><<<CAFFE_GET_BLOCKS(ndata), CAFFE_CUDA_NUM_THREADS>>>(ndata, start_index, input_data, target, PP, GG, PG, num*channel, bottom_diff);
	    } else {
	      DiceLossNewBackward<Dtype><<<CAFFE_GET_BLOCKS(ndata), CAFFE_CUDA_NUM_THREADS>>>(ndata, start_index, input_data, target, PP, GG, PG, 1, bottom_diff);
	    }
          }
        }
      }
    }
  }

INSTANTIATE_LAYER_GPU_FUNCS(DiceLossNewLayer);

}  // namespace caffe
